#include "hip/hip_runtime.h"
// Copyright (c) 2018-2020, Michael P. Howard
// Copyright (c) 2021-2024, Auburn University
// Part of azplugins, released under the BSD 3-Clause License.

#include "DihedralBendingTorsionForceGPU.cuh"
#include "hoomd/TextureTools.h"

#include <assert.h>

/*! \file DihedralBendingTorsionForceGPU.cu
    \brief Defines GPU kernel code for calculating Bending-Torsion dihedral forces. Used by
   DihedralBendingTorsionForceComputeGPU.
*/

namespace hoomd
    {
namespace azplugins
    {
namespace gpu
    {
//! Kernel for calculating combined bending-torsion dihedral forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the device
    \param d_params Array of combined B-T parameters k_phi, a0,a1,a2,a3,a4
    \param box Box dimensions for periodic boundary condition handling
    \param tlist Dihedral data to use in calculating the forces
    \param dihedral_ABCD List of relative atom positions in the dihedrals
    \param pitch Pitch of 2D dihedral list
    \param n_dihedrals_list List of numbers of dihedrals per atom
*/
__global__ void gpu_compute_bending_torsion_dihedral_forces_kernel(Scalar4* d_force,
                                                        Scalar* d_virial,
                                                        const size_t virial_pitch,
                                                        const unsigned int N,
                                                        const Scalar4* d_pos,
                                                        const dihedral_bending_torsion_params* d_params,
                                                        BoxDim box,
                                                        const group_storage<4>* tlist,
                                                        const unsigned int* dihedral_ABCD,
                                                        const unsigned int pitch,
                                                        const unsigned int* n_dihedrals_list)
    {
    // start by identifying which particle we are to handle
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_dihedrals = n_dihedrals_list[idx];

    // read in the position of our b-particle from the a-b-c-d set. (MEM TRANSFER: 16 bytes)
    Scalar4 idx_postype = d_pos[idx]; // we can be either a, b, or c in the a-b-c-d quartet
    Scalar3 idx_pos = make_scalar3(idx_postype.x, idx_postype.y, idx_postype.z);
    Scalar3 pos_a, pos_b, pos_c,
        pos_d; // allocate space for the a,b, and c atoms in the a-b-c-d quartet

    // initialize the force to 0
    Scalar4 force_idx = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));

    // initialize the virial to 0
    Scalar virial_idx[6];
    for (unsigned int i = 0; i < 6; i++)
        virial_idx[i] = Scalar(0.0);

    // loop over all dihedrals
    for (int dihedral_idx = 0; dihedral_idx < n_dihedrals; dihedral_idx++)
        {
        group_storage<4> cur_dihedral = tlist[pitch * dihedral_idx + idx];
        unsigned int cur_ABCD = dihedral_ABCD[pitch * dihedral_idx + idx];

        int cur_dihedral_x_idx = cur_dihedral.idx[0];
        int cur_dihedral_y_idx = cur_dihedral.idx[1];
        int cur_dihedral_z_idx = cur_dihedral.idx[2];
        int cur_dihedral_type = cur_dihedral.idx[3];
        int cur_dihedral_abcd = cur_ABCD;

        // get the a-particle's position (MEM TRANSFER: 16 bytes)
        Scalar4 x_postype = d_pos[cur_dihedral_x_idx];
        Scalar3 x_pos = make_scalar3(x_postype.x, x_postype.y, x_postype.z);
        // get the c-particle's position (MEM TRANSFER: 16 bytes)
        Scalar4 y_postype = d_pos[cur_dihedral_y_idx];
        Scalar3 y_pos = make_scalar3(y_postype.x, y_postype.y, y_postype.z);
        // get the c-particle's position (MEM TRANSFER: 16 bytes)
        Scalar4 z_postype = d_pos[cur_dihedral_z_idx];
        Scalar3 z_pos = make_scalar3(z_postype.x, z_postype.y, z_postype.z);

        if (cur_dihedral_abcd == 0)
            {
            pos_a = idx_pos;
            pos_b = x_pos;
            pos_c = y_pos;
            pos_d = z_pos;
            }
        if (cur_dihedral_abcd == 1)
            {
            pos_b = idx_pos;
            pos_a = x_pos;
            pos_c = y_pos;
            pos_d = z_pos;
            }
        if (cur_dihedral_abcd == 2)
            {
            pos_c = idx_pos;
            pos_a = x_pos;
            pos_b = y_pos;
            pos_d = z_pos;
            }
        if (cur_dihedral_abcd == 3)
            {
            pos_d = idx_pos;
            pos_a = x_pos;
            pos_b = y_pos;
            pos_c = z_pos;
            }

        // the three bonds

        Scalar3 vb1 = pos_a - pos_b;
        Scalar3 vb2 = pos_c - pos_b;
        Scalar3 vb3 = pos_d - pos_c;

        // apply periodic boundary conditions
        vb1 = box.minImage(vb1);
        vb2 = box.minImage(vb2);
        vb3 = box.minImage(vb3);

        Scalar3 vb2m = -vb2;
        vb2m = box.minImage(vb2m);

        // c,s calculation

        Scalar ax, ay, az, bx, by, bz;
        ax = vb1.y * vb2m.z - vb1.z * vb2m.y;
        ay = vb1.z * vb2m.x - vb1.x * vb2m.z;
        az = vb1.x * vb2m.y - vb1.y * vb2m.x;
        bx = vb3.y * vb2m.z - vb3.z * vb2m.y;
        by = vb3.z * vb2m.x - vb3.x * vb2m.z;
        bz = vb3.x * vb2m.y - vb3.y * vb2m.x;

        Scalar rasq = ax * ax + ay * ay + az * az;
        Scalar rbsq = bx * bx + by * by + bz * bz;
        Scalar rgsq = vb2m.x * vb2m.x + vb2m.y * vb2m.y + vb2m.z * vb2m.z;
        Scalar rg = fast::sqrt(rgsq);

        Scalar rginv, ra2inv, rb2inv;
        rginv = ra2inv = rb2inv = 0.0;
        if (rg > 0)
            rginv = 1.0 / rg;
        if (rasq > 0)
            ra2inv = 1.0 / rasq;
        if (rbsq > 0)
            rb2inv = 1.0 / rbsq;
        Scalar rabinv = fast::sqrt(ra2inv * rb2inv);

        Scalar c = (ax * bx + ay * by + az * bz) * rabinv;
        Scalar s = rg * rabinv * (ax * vb3.x + ay * vb3.y + az * vb3.z);

        if (c > 1.0)
            c = 1.0;
        if (c < -1.0)
            c = -1.0;

        // get values for k1/2 through k4/2 (MEM TRANSFER: 16 bytes)
        // ----- The 1/2 factor is already stored in the parameters --------
        /**
         * 
        dihedral_bending_torsion_params params = __ldg(d_params + cur_dihedral_type);
        Scalar k_phi = h_params.data[dihedral_type].k_phi;
        Scalar a0 = h_params.data[dihedral_type].a0;
        Scalar a1 = h_params.data[dihedral_type].a1;
        Scalar a2 = h_params.data[dihedral_type].a2;
        Scalar a3 = h_params.data[dihedral_type].a3;
        Scalar a4 = h_params.data[dihedral_type].a4;
         */
        dihedral_bending_torsion_params params = d_params[cur_dihedral_type];
        Scalar k_phi = params.k_phi;
        Scalar a0 = params.a0;
        Scalar a1 = params.a1;
        Scalar a2 = params.a2;
        Scalar a3 = params.a3;
        Scalar a4 = params.a4;

        /**

        fg = vb1.x * vb2m.x + vb1.y * vb2m.y + vb1.z * vb2m.z;
        hg = vb3.x * vb2m.x + vb3.y * vb2m.y + vb3.z * vb2m.z;
        fga = fg * ra2inv * rginv;
        hgb = hg * rb2inv * rginv;
        gaa = -ra2inv * rg;
        gbb = rb2inv * rg;

        dtfx = gaa * ax;
        dtfy = gaa * ay;
        dtfz = gaa * az;
        dtgx = fga * ax - hgb * bx;
        dtgy = fga * ay - hgb * by;
        dtgz = fga * az - hgb * bz;
        dthx = gbb * bx;
        dthy = gbb * by;
        dthz = gbb * bz;

        sx2 = df * dtgx;
        sy2 = df * dtgy;
        sz2 = df * dtgz;

        f1.x = df * dtfx;
        f1.y = df * dtfy;
        f1.z = df * dtfz;
        f1.w = e_dihedral;

        f2.x = sx2 - f1.x;
        f2.y = sy2 - f1.y;
        f2.z = sz2 - f1.z;
        f2.w = e_dihedral;

        f4.x = df * dthx;
        f4.y = df * dthy;
        f4.z = df * dthz;
        f4.w = e_dihedral;

        f3.x = -sx2 - f4.x;
        f3.y = -sy2 - f4.y;
        f3.z = -sz2 - f4.z;
        f3.w = e_dihedral;

        // Apply force to each of the 4 atoms
        h_force.data[i1].x = h_force.data[i1].x + f1.x;
        h_force.data[i1].y = h_force.data[i1].y + f1.y;
        h_force.data[i1].z = h_force.data[i1].z + f1.z;
        h_force.data[i1].w = h_force.data[i1].w + f1.w;
        h_force.data[i2].x = h_force.data[i2].x + f2.x;
        h_force.data[i2].y = h_force.data[i2].y + f2.y;
        h_force.data[i2].z = h_force.data[i2].z + f2.z;
        h_force.data[i2].w = h_force.data[i2].w + f2.w;
        h_force.data[i3].x = h_force.data[i3].x + f3.x;
        h_force.data[i3].y = h_force.data[i3].y + f3.y;
        h_force.data[i3].z = h_force.data[i3].z + f3.z;
        h_force.data[i3].w = h_force.data[i3].w + f3.w;
        h_force.data[i4].x = h_force.data[i4].x + f4.x;
        h_force.data[i4].y = h_force.data[i4].y + f4.y;
        h_force.data[i4].z = h_force.data[i4].z + f4.z;
        h_force.data[i4].w = h_force.data[i4].w + f4.w;

        // Compute 1/4 of the virial, 1/4 for each atom in the dihedral
        // upper triangular version of virial tensor
        dihedral_virial[0] = 0.25 * (vb1.x * f1.x + vb2.x * f3.x + (vb3.x + vb2.x) * f4.x);
        dihedral_virial[1] = 0.25 * (vb1.y * f1.x + vb2.y * f3.x + (vb3.y + vb2.y) * f4.x);
        dihedral_virial[2] = 0.25 * (vb1.z * f1.x + vb2.z * f3.x + (vb3.z + vb2.z) * f4.x);
        dihedral_virial[3] = 0.25 * (vb1.y * f1.y + vb2.y * f3.y + (vb3.y + vb2.y) * f4.y);
        dihedral_virial[4] = 0.25 * (vb1.z * f1.y + vb2.z * f3.y + (vb3.z + vb2.z) * f4.y);
        dihedral_virial[5] = 0.25 * (vb1.z * f1.z + vb2.z * f3.z + (vb3.z + vb2.z) * f4.z);

        for (int k = 0; k < 6; k++)
            {
            h_virial.data[virial_pitch * k + i1] += dihedral_virial[k];
            h_virial.data[virial_pitch * k + i2] += dihedral_virial[k];
            h_virial.data[virial_pitch * k + i3] += dihedral_virial[k];
            h_virial.data[virial_pitch * k + i4] += dihedral_virial[k];
            }
        */
        }

    // // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    // d_force[idx] = force_idx;
    // for (int k = 0; k < 6; k++)
    //     d_virial[k * virial_pitch + idx] = virial_idx[k];
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the GPU
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param tlist Dihedral data to use in calculating the forces
    \param dihedral_ABCD List of relative atom positions in the dihedrals
    \param pitch Pitch of 2D dihedral list
    \param n_dihedrals_list List of numbers of dihedrals per atom
    \param d_params Array of combined B-T parameters k1/2, k2/2, k3/2, and k4/2
    \param n_dihedral_types Number of dihedral types in d_params
    \param block_size Block size to use when performing calculations
    \param compute_capability Compute capability of the device (200, 300, 350, ...)

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one Scalar4 element per dihedral type. The x component contains K the
   spring constant and the y component contains sign, and the z component the multiplicity.
*/
hipError_t gpu_compute_bending_torsion_dihedral_forces(Scalar4* d_force,
                                            Scalar* d_virial,
                                            const size_t virial_pitch,
                                            const unsigned int N,
                                            const Scalar4* d_pos,
                                            const BoxDim& box,
                                            const group_storage<4>* tlist,
                                            const unsigned int* dihedral_ABCD,
                                            const unsigned int pitch,
                                            const unsigned int* n_dihedrals_list,
                                            dihedral_bending_torsion_params* d_params,
                                            const unsigned int n_dihedral_types,
                                            const int block_size,
                                            const int warp_size)
    {
    assert(d_params);

    unsigned int max_block_size;
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))gpu_compute_bending_torsion_dihedral_forces_kernel);
    max_block_size = attr.maxThreadsPerBlock;
    if (max_block_size % warp_size)
        // handle non-sensical return values from hipFuncGetAttributes
        max_block_size = (max_block_size / warp_size - 1) * warp_size;

    unsigned int run_block_size = min(block_size, max_block_size);

    // setup the grid to run the kernel
    dim3 grid(N / run_block_size + 1, 1, 1);
    dim3 threads(run_block_size, 1, 1);

    // run the kernel
    hipLaunchKernelGGL((gpu_compute_bending_torsion_dihedral_forces_kernel),
                       dim3(grid),
                       dim3(threads),
                       0,
                       0,
                       d_force,
                       d_virial,
                       virial_pitch,
                       N,
                       d_pos,
                       d_params,
                       box,
                       tlist,
                       dihedral_ABCD,
                       pitch,
                       n_dihedrals_list);

    return hipSuccess;
    }

    } // end namespace gpu
    } // end namespace azplugins
    } // end namespace hoomd
